#include "hip/hip_runtime.h"
#include "manogpu.h"

#include <cstdio>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hipblas.h>

__global__
void compute_normals_kernel(float *vertices, float *normals, GLuint *elements, int triangleCount) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= triangleCount) {
        return;
    }
    int ai = 3*elements[3*i+0];
    int bi = 3*elements[3*i+1];
    int ci = 3*elements[3*i+2];

    float *a = &vertices[ai];
    float *b = &vertices[bi];
    float *c = &vertices[ci];

    float x[3] = {b[0]-a[0], b[1]-a[1], b[2]-a[2]};
    float y[3] = {c[0]-a[0], c[1]-a[1], c[2]-a[2]};

    float p[3];
    p[0] = x[1]*y[2]-y[1]*x[2];
    p[1] = x[2]*y[0]-y[2]*x[0];
    p[2] = x[0]*y[1]-y[0]*x[1];

    atomicAdd(normals+ai+0, p[0]);
    atomicAdd(normals+ai+1, p[1]);
    atomicAdd(normals+ai+2, p[2]);

    atomicAdd(normals+bi+0, p[0]);
    atomicAdd(normals+bi+1, p[1]);
    atomicAdd(normals+bi+2, p[2]);

    atomicAdd(normals+ci+0, p[0]);
    atomicAdd(normals+ci+1, p[1]);
    atomicAdd(normals+ci+2, p[2]);
}

__global__
void normalize_normals_kernel(float *normals, int vertexCount) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= vertexCount) {
        return;
    }
    const int offset = 3*i;
    const float x = normals[offset+0];
    const float y = normals[offset+1];
    const float z = normals[offset+2];
    const float len = sqrtf(x*x+y*y+z*z);
//    const float ilen = (len>0.001)?1.0f/len:0.0f;
    const float ilen = 1/len;
    normals[offset+0] *= ilen;
    normals[offset+1] *= ilen;
    normals[offset+2] *= ilen;
}

__global__
void zero_normals_kernel(float *normals, int vertexCount) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= vertexCount) {
        return;
    }
    const int offset = 3*i;
    normals[offset+0] = 0.0f;
    normals[offset+1] = 0.0f;
    normals[offset+2] = 0.0f;
}

void compute_normals_gpu(float *gpuVertices, float *gpuNormals, GLuint *gpuElements, int vertexCount, int triangleCount) {
    const dim3 threadsPerBlock(32, 1);
    {
        const dim3 blocksPerGrid((vertexCount + threadsPerBlock.x - 1) / threadsPerBlock.x, 1);
        zero_normals_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpuNormals, vertexCount);
    }
    {
        const dim3 blocksPerGrid((triangleCount + threadsPerBlock.x - 1) / threadsPerBlock.x, 1);
        compute_normals_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpuVertices, gpuNormals, gpuElements, triangleCount);
    }
    {
        const dim3 blocksPerGrid((vertexCount + threadsPerBlock.x - 1) / threadsPerBlock.x, 1);
        normalize_normals_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpuNormals, vertexCount);
    }
}

//void compute_normals(float *vertices, GLuint *elements, int vertexCount, int triangleCount) {
//    float *gpuVertices;
//    GLuint *gpuElements;
//    hipMalloc(&gpuVertices, 6*sizeof(float)*vertexCount);
//    hipMalloc(&gpuElements, 3*sizeof(GLuint)*triangleCount);

//    hipMemcpy(gpuVertices, vertices, 6*sizeof(float)*vertexCount, hipMemcpyHostToDevice);
//    hipMemcpy(gpuElements, elements, 3*sizeof(GLuint)*triangleCount, hipMemcpyHostToDevice);

//    compute_normals_gpu(gpuVertices, gpuElements, vertexCount, triangleCount);

//    hipMemcpy(vertices, gpuVertices, 6*sizeof(float)*vertexCount, hipMemcpyDeviceToHost);

//    hipFree(gpuVertices);
//    hipFree(gpuElements);
//}

void init_mano_gpu(manoData_t *data, GLuint vbo, GLuint nbo, GLuint ebo, float *vShaped, float *poseDirs, int vertexCount, int triangleCount) {
    data->vertexCount = vertexCount;
    data->triangleCount = triangleCount;

    data->vbo = vbo;
    hipGraphicsGLRegisterBuffer((hipGraphicsResource**)&data->cudaVboResource, data->vbo, cudaGraphicsMapFlagsWriteDiscard);

    data->nbo = nbo;
    hipGraphicsGLRegisterBuffer((hipGraphicsResource**)&data->cudaNboResource, data->nbo, cudaGraphicsMapFlagsWriteDiscard);

    data->ebo = ebo;
    hipGraphicsGLRegisterBuffer((hipGraphicsResource**)&data->cudaEboResource, data->ebo, cudaGraphicsMapFlagsReadOnly);

    hipMalloc(&data->cudaVShaped, data->vertexCount*3*sizeof(float));
    hipMalloc(&data->cudaPoseDirs, data->vertexCount*3*459*sizeof(float));
    hipMalloc(&data->cudaPosemapped, 459*sizeof(float));

    hipMemcpy(data->cudaVShaped, vShaped, data->vertexCount*3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(data->cudaPoseDirs, poseDirs, data->vertexCount*3*459*sizeof(float), hipMemcpyHostToDevice);

    data->cublasHandle = (void*)new hipblasHandle_t;
    hipblasCreate((hipblasHandle_t*)data->cublasHandle);
}

void destroy_mano_gpu(manoData_t *data) {
    hipblasDestroy(*(hipblasHandle_t*)data->cublasHandle);
    delete (hipblasHandle_t*)data->cublasHandle;

    hipFree(data->cudaPoseDirs);
    hipFree(data->cudaPosemapped);

    hipGraphicsUnregisterResource((hipGraphicsResource*)data->cudaVboResource);
    hipGraphicsUnregisterResource((hipGraphicsResource*)data->cudaNboResource);
    hipGraphicsUnregisterResource((hipGraphicsResource*)data->cudaEboResource);
}

void update_mano_vshaped(manoData_t *data, float *vShaped) {
    hipMemcpy(data->cudaVShaped, vShaped, data->vertexCount*3*sizeof(float), hipMemcpyHostToDevice);
}

__global__
void mult_add(float *A, float *x, float *output, int rows, int cols) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= rows) {
        return;
    }

    float res = output[i];
    for (int j = 0; j < cols; ++j) {
        res += A[i*cols+j]*x[j];
    }
    output[i] = res;
}

void compute_mano_and_normals_vbo(manoData_t *data) {
    hipGraphicsResource *resources[] = {
        (hipGraphicsResource*)data->cudaVboResource,
        (hipGraphicsResource*)data->cudaNboResource,
        (hipGraphicsResource*)data->cudaEboResource
    };
    hipGraphicsMapResources(3, resources, 0);

    float *vertices, *normals;
    GLuint *elements;

    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void**)&vertices, &num_bytes, resources[0]);
    hipGraphicsResourceGetMappedPointer((void**)&normals, &num_bytes, resources[1]);
    hipGraphicsResourceGetMappedPointer((void**)&elements, &num_bytes, resources[2]);

    hipMemcpy(vertices, data->cudaVShaped, sizeof(float)*data->vertexCount*3, hipMemcpyDeviceToDevice);
    hipMemcpy(data->cudaPosemapped, data->hostPosemapped, sizeof(float)*459, hipMemcpyHostToDevice);

//    const int tpb = 32;
//    const int bpg = (data->vertexCount*3+31)/tpb;
//    mult_add<<<bpg, tpb>>>(data->cudaPoseDirs, data->cudaPosemapped, vertices, data->vertexCount*3, 459);
    float one = 1.0f;
    float deform_weight = 1.0f;
    hipblasSgemv(*(hipblasHandle_t*)data->cublasHandle, HIPBLAS_OP_T,
                459, data->vertexCount*3, &deform_weight, data->cudaPoseDirs, 459,
                data->cudaPosemapped, 1, &one, vertices, 1);

    compute_normals_gpu(vertices, normals, elements, data->vertexCount, data->triangleCount);

    hipGraphicsUnmapResources(3, resources, 0);
}
